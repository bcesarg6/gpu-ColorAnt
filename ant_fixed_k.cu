#include "hip/hip_runtime.h"
/***********************************************************
 * Created: Seg 10 Out 2011 17:17:12 BRT
 *
 * Author: Carla N. Lintzmayer, carla0negri@gmail.com
 *
 * Modificado em 2017
 * Autor: Bruno Cesar Puli Dala Rosa
 *
 * ANT_FIXED_K
 * * Constructive method for each ant in an ACO algorithm for k-GCP
 *
 ***********************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <limits.h>

#include "color.h"
#include "aco.h"
#include "ant_fixed_k.h"
#include "util.h"

/* ANT_FIXED_K data */
static double *probb;
static double **trail;
static float alpha;
static float beta;

static int choose_vertex(int **neighbors_by_color, int *color_of) {/*{{{*/

  int v = 0, i, dsat, maxdsat;

  /* Choose vertex with maximum degree of saturation */
  maxdsat = -1;
  for (i = 0; i < problem->nof_vertices; i++) {
    if (color_of[i] == -1) {
      dsat = neighbors_by_color[i][problem->max_colors];
      //printf("dsat : %d\n", dsat);
      if (dsat > maxdsat) {
         //printf("Entrou dsat : %d\n", dsat);

	v = i;
	maxdsat = dsat;
      }
    }
  }
  return v;

}/*}}}*/

static void calculate_probbs(int v, int *color_of, int *size_color, int **neighbors_by_color) { /*{{{*/

  int c;
  double sum, traill, totalsum, neighbors;

  totalsum = 0;

  for (c = 0; c < problem->max_colors; c++) {

    probb[c] = 0;

    sum = trail[c][v];

#if defined COLORANT
    if (aco_info->colorid == COLORANT4) {
      if (size_color[c] == 0) {
        traill = aco_info->y;
      }
      else {
        if (neighbors_by_color[v][c] == 0) {
          traill = aco_info->x;
        }
        else {
          traill = sum/size_color[c];
        }
      }
    }
    else
#endif
      traill = (size_color[c] == 0) ? 1 : sum/size_color[c];

    neighbors = neighbors_by_color[v][c] + 1;
    neighbors = 1.0/neighbors;

    probb[c] = pow(traill, alpha) * pow(neighbors, beta);

    totalsum += probb[c];
  }

  totalsum = (totalsum == 0) ? 1 : totalsum;
  /* To avoid a new 'for'  */
  probb[problem->max_colors] = totalsum;

}/*}}}*/

static int choose_color(void) {/*{{{*/

  int i;
  double p, last, div;

  div = probb[problem->max_colors];
  //printf(" div : %lf ,", div);

#if defined LRAND
  //p = (double) RANDOM_UNIT() / INT_MAX;
  RANDOM_UNIT(problem->buffer, p, double);
  p = p / INT_MAX;
#elif defined NRAND
  //p = (double) RANDOM_UNIT(problem->seed) / INT_MAX;
  RANDOM_UNIT(problem->seed, problem->buffer, p, double);
  p = p / INT_MAX;
#endif
  last = 0;
 // printf(" rand : %lf , ", p );
  for (i = 0; i < problem->max_colors; i++) {
    last += (probb[i]/div);
    if (p <= last){
      return i;
    }
  }
  /* When it reaches here, it means that p == 1 */
  return problem->max_colors -1;

}/*}}}*/


void afk_initialize_data(float p_alpha, float p_beta) {/*{{{*/

  int i, j;

  probb = (double*) malloc(sizeof(double) * (problem->max_colors + 1));
  trail = (double**) malloc(sizeof(double*) * problem->max_colors);

  for (i = 0; i < problem->max_colors; i++) {
    trail[i] = (double*) malloc(sizeof(double) * problem->nof_vertices);
    for (j = 0; j < problem->nof_vertices; j++) {
      trail[i][j] = 0;
    }
  }

  alpha = p_alpha;
  beta = p_beta;
}/*}}}*/

void ant_fixed_k(gcp_solution_t *solution, double **pheromone) {/*{{{*/

  int i, j;
  int color = 0;			/* number of colors to be used */
  int colored = 0;		/* number of colored vertex */
  int v;					/* vertex to be colored */

  int *confl_vertices;
  int **neighbors_by_color;
  int *size_color;
  solution->nof_colors = problem->max_colors;

  size_color = (int*) malloc(sizeof(int) * problem->max_colors);
  confl_vertices = (int*) malloc(sizeof(int) * problem->nof_vertices);
  neighbors_by_color = (int**) malloc(sizeof(int*) * problem->nof_vertices);

  for (i = 0; i < problem->nof_vertices; i++){
      neighbors_by_color[i] = (int*) malloc(sizeof(int) * problem->max_colors+1);
  }

  /* Initializing auxiliary arrays */
  for (i = 0; i < problem->nof_vertices; i++) {
    solution->color_of[i] = -1;
    size_color[i] = 0;
    confl_vertices[i] = 0;
    for (j = 0; j < problem->max_colors; j++) {
      neighbors_by_color[i][j] = 0;
      trail[j][i] = 0;
    }
    neighbors_by_color[i][problem->max_colors] = 0;
  }

  solution->nof_confl_edges = 0;
  solution->nof_confl_vertices = 0;

 // printf("\nStart\n\n");
  while (colored < problem->nof_vertices) {

    /* Chose a vertex to be colored */
    v = choose_vertex(neighbors_by_color, solution->color_of);
    //printf("v : %d , ", v);
    /* Calculate colors probabilities */
    calculate_probbs(v, solution->color_of, size_color, neighbors_by_color);

    /* Choose a color to be assigned to v */
    color = choose_color();
    //printf("c : %d \n", color);

    solution->color_of[v] = color;
    size_color[color]++;
    colored++;

    /* Update informations about conflicts and saturation degree */
    int conf = solution->nof_confl_edges;
    for (i = 0; i < problem->nof_vertices; i++) {

      /* trail keeps the pheromone between a vertex and all the vertex
       * already colored with each color */
      trail[color][i] += pheromone[v][i];

      if (problem->adj_matrix[v * problem->nof_vertices + i]) {
	/* update degree of saturation: */
	if (neighbors_by_color[i][color] == 0) {
	  neighbors_by_color[i][problem->max_colors]++;
	}
	/* now <i> has a neighbor colored with <color> */
	neighbors_by_color[i][color]++;

	/* if a neighbor of <v> is colored with <color>, there is a
	 * conflicting edge between them */
	if (solution->color_of[i] == color) {
	  solution->nof_confl_edges++;
	  if (confl_vertices[i] == 0) {
	    confl_vertices[i] = 1;
	    solution->nof_confl_vertices++;
	  }
	}
      }
    }
    /* if any new conflicting edge was created, <v> is a conflicting
     * vertex */
    if (conf != solution->nof_confl_edges) {
      if (confl_vertices[v] == 0) {
	confl_vertices[v] = 1;
	solution->nof_confl_vertices++;
      }
    }
  }

  solution->spent_time = current_time_secs(TIME_FINAL, time_initial);

}/*}}}*/

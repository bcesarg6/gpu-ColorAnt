#include "hip/hip_runtime.h"
/***********************************************************
 * Created: 19 jun 2017
 *
 * Author: Bruno C.P. Dala Rosa, bcesar.g6@gmail.com
 *
 * GPU_ANT_FIXED_K_B
 * * Parallel Constructive method for n-ants in an ACO algorithm for k-GCP
 *
 ***********************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <limits.h>

/* CUDA runtime */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
/* CUDA cuRand */
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>

#include "color.h"
#include "aco.h"
#include "util.h"
#include "gpu_ant_fixed_k.cuh"

double *probb;
double *probb_totalsum;
int *vertices;				  /* vertices a serem coloridos*/
int *colors;                  /* cores a serem atribuidas aos respectivo vertices */
int *solutions_color_of;
int *nof_confl_edges;
int *nof_confl_vertices;

/* Device pointers */
//int* d_nof_confl_vertices;
//int* d_solutions_color_of;
int* d_vertices;
int* d_colors;
int* d_vertices_sat;
int* d_neighbors_by_color;
int* d_size_color;
double* d_trail;
double* d_probb;
double* d_probb_totalsum;
int* d_nof_confl_edges;
int* d_confl_vertices;
int* d_conf;

/* Constant device memory */
__constant__ aco_t d_aco_info;

int ants;

/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/* Transfer Constant memory to device */
__host__ void copiaConstant(gcp_t* problem, aco_t* aco_info){
    gpuErrchk(hipMemcpy(d_problem, problem, sizeof(gcp_t), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_aco_info), aco_info, sizeof(aco_t)));
}

__global__ void setup_cuRand (hiprandState* states, unsigned long seed){
  hiprand_init ( seed, blockIdx.x , 0, &states[blockIdx.x] );
}

/* Choose vertex by Reducing using shared memory */
__global__ void choose_vertex(gcp_t* d_problem, int* d_vertices, int* d_vertices_sat, int* d_solutions_color_of){
	int tid = threadIdx.x;

	/*Vertices ceil values*/
	extern __shared__ short s[];
	short* s_indexes = s;
	short* s_values = &s_indexes[d_problem->ceil_vertices];
    short* s_color_aux = &s_indexes[d_problem->ceil_vertices * 2];

	s_indexes[tid] = tid;
    if(tid < d_problem->nof_vertices){
        s_values[tid] = d_vertices_sat[threadIdx.x + d_problem->nof_vertices * blockIdx.x];
        s_color_aux[tid] = d_solutions_color_of[threadIdx.x + d_problem->nof_vertices * blockIdx.x];
    } else{
        s_color_aux[tid] = 1;
    }

	__syncthreads();

	if (s_color_aux[tid] >= 0 || tid > d_problem->nof_vertices){
        //printf("color_aux = %d\t", d_color_of_aux[globalID]);
		s_values[tid] = -1;
	}
	__syncthreads();

	//do reduction in shared memory
	for (unsigned int s = blockDim.x / 2; s > 0; s >>=1){
		if (tid < s){
			if ((s_values[tid] < s_values[tid + s]) || (s_values[tid] == s_values[tid + s]) && (s_indexes[tid] > s_indexes[tid + s])){
                //printf("Index %d[%d] = %d[%d]\n",tid, s_values[tid], tid+s, s_values[tid+s]);
				s_values[tid] = s_values[tid + s];
				s_indexes[tid] = s_indexes[tid + s];
			}
		}
		__syncthreads();
	}

	//only thread 0 writes result for this block back to global memory
	if (tid == 0){
		d_vertices[blockIdx.x] = s_indexes[0];
	}
}

__global__ void calculate_probbs(gcp_t* d_problem, double* d_probb, double* d_trail, int* d_size_color, int* d_neighbors_by_color, int* d_vertices, double *d_probb_totalsum){
      int threadID = threadIdx.x;
      int blockID = blockIdx.x;
      double sum, traill, neighbors;
      int size_color, neighbor;

      neighbor   = d_neighbors_by_color[(blockID * d_problem->nof_vertices * d_problem->max_colors) + (d_vertices[blockID] * d_problem->max_colors) + threadID];
      size_color = d_size_color[(blockID *  d_problem->max_colors) + threadID];
      sum        = d_trail[(blockID * d_problem->max_colors * d_problem->nof_vertices) + (threadID * d_problem->nof_vertices) + d_vertices[blockID]]; //Acesso direto a global memory por que só é lido uma vez

      #if defined COLORANT
          if (get_flag(d_problem->flags, FLAG_REUSE_COLOR)) {
              if (size_color == 0) {
                  traill = d_aco_info.y;
              } else {
                  if (neighbor == 0) {
                      traill = d_aco_info.x;
                  } else {
                      traill = sum/size_color;
                  }
              }
          }
      #endif

      traill = (size_color == 0) ? 1 : sum/size_color;
      neighbors = neighbor + 1;
      neighbors = 1.0/neighbors;
      d_probb[(blockID * d_problem->max_colors) + threadID] = pow(traill, (double)d_aco_info.alpha) * pow(neighbors, (double)d_aco_info.beta);
    }

__global__ void probbs_sum(gcp_t* d_problem, double* d_probb, double* d_probb_totalsum){
    int tid = threadIdx.x;
    int max_colors = d_problem->max_colors;

    extern __shared__ double sh[];
    double* s_probb = sh;

    if(tid < max_colors){
        s_probb[tid] = d_probb[tid + max_colors * blockIdx.x];
    } else{
        s_probb[tid] = 0;
    }
    __syncthreads();

    //do reduction in shared memory
    for (unsigned int i = blockDim.x / 2; i > 0; i >>=1){
        if (tid < i && i < max_colors){
            s_probb[tid] += s_probb[tid + i];
        }
        __syncthreads();
    }

    //only thread 0 writes result for this block back to global memory
    if (tid == 0){
        d_probb_totalsum[blockIdx.x] = s_probb[0];
    }
}


/* Usando shared memory e apenas uma thread per block, comparar o desempenho com abordagem global memory e n threads per block */
__global__ void choose_color(gcp_t* d_problem, int *d_colors, hiprandState_t *states, double *d_probb_totalsum, double *d_probb){
    int threadID = threadIdx.x;
    int blockID = blockIdx.x;
    int i,v;
    double p, last, div;
    last = 0;
    v = 1;

    /* Shared memory setup */
    extern __shared__ double sh[];
    double *s_probb = sh;
    s_probb[threadID] = d_probb[blockID * d_problem->max_colors + threadID];
    __syncthreads();
    /*---------------------*/

    div = d_probb_totalsum[blockID];

    if (threadID == 0){
      p = hiprand_uniform_double(&states[blockID]);
      //printf("id:%d -> %lf - %lf\n", blockID, div, p); //debugging
      for (i = 0; i < d_problem->max_colors; i++) {
        last += (s_probb[i]/div);
        if (p <= last) {
          v = 0;
          d_colors[blockID] = i;
          i = d_problem->max_colors;
        }
      }
      /* When it reaches here, it means that p == 1 */
      if(v) d_colors[blockID] = d_problem->max_colors -1;
    }
}

__global__ void sol_colors_reset(int* d_solutions_color_of){
    int globalID = threadIdx.x + blockDim.x * blockIdx.x;
    d_solutions_color_of[globalID] = -1;
}

/*Reseta os dados do device que precisam ser resetados */
__host__ void gpu_ant_fixed_k_reset(gcp_t* d_problem){
    sol_colors_reset<<<ants,problem->nof_vertices>>>(d_solutions_color_of);

    gpuErrchk(hipMemset( d_vertices_sat, 0, sizeof(int) * ants * problem->nof_vertices));
    gpuErrchk(hipMemset( d_vertices, 0, sizeof(int) * ants));
    gpuErrchk(hipMemset( d_colors, 0, sizeof(int) * ants));
    gpuErrchk(hipMemset( d_neighbors_by_color, 0, sizeof(int) * ants * problem->nof_vertices * problem->max_colors));
    gpuErrchk(hipMemset( d_size_color, 0, sizeof(int) * ants * problem->max_colors));
    gpuErrchk(hipMemset( d_trail, 0, sizeof(double) * ants * problem->nof_vertices * problem->max_colors));
    gpuErrchk(hipMemset( d_probb, 0, sizeof(double) * ants * problem->max_colors));
    gpuErrchk(hipMemset( d_probb_totalsum, 0, sizeof(double) * ants));
    gpuErrchk(hipMemset( d_nof_confl_edges, 0, sizeof(int) * ants));
    gpuErrchk(hipMemset( d_nof_confl_vertices, 0, sizeof(int) * ants));
    gpuErrchk(hipMemset( d_confl_vertices, 0, sizeof(int) * ants * problem->nof_vertices));

    //Host
    memset( vertices, 0, sizeof(int) * ants);
    memset( colors, 0, sizeof(int) * ants);
}

__host__ void ant_fixed_initilization(gcp_t* d_problem){
    ants = aco_info->n_threads;

    probb = (double*) malloc(sizeof(double) * ants * problem->max_colors);                                 //[ants][max_colors]
    probb_totalsum = (double*) malloc(sizeof(double) * ants);                                              //[ants]
    vertices = (int*) malloc(sizeof(int) * ants);                                                          //[ants]
    colors = (int*) malloc(sizeof(int) * ants);                                                            //[ants]
    solutions_color_of = (int*) malloc(sizeof(int) * ants * problem->nof_vertices);                        //[ants][nof_vertices]
    nof_confl_edges = (int*) malloc(sizeof(int) * ants);                                                   //[ants]
    nof_confl_vertices = (int*) malloc(sizeof(int) * ants);                                                //[ants]

    gpuErrchk(hipMalloc((void **) &d_vertices, sizeof(int) * ants));                                                          //[ants]
    gpuErrchk(hipMalloc((void **) &d_colors, sizeof(int) * ants));                                                            //[ants]
    gpuErrchk(hipMalloc((void **) &d_vertices_sat, sizeof(int) * ants * problem->nof_vertices));                              //[ants][nof_vertices] <-> 1024 Testando abordagem de arredondamento!!
    gpuErrchk(hipMalloc((void **) &d_solutions_color_of, sizeof(int) * ants * problem->nof_vertices));                        //[ants][nof_vertices]; <-> 1024 Testando abordagem de arredondamento!!
    gpuErrchk(hipMalloc((void **) &d_neighbors_by_color, sizeof(int) * ants * problem->nof_vertices * problem->max_colors));  //[ants][nof_vertices][max_colors]
    gpuErrchk(hipMalloc((void **) &d_size_color, sizeof(int) * ants * problem->max_colors));                                  //[ants][max_colors]
    gpuErrchk(hipMalloc((void **) &d_trail, sizeof(double) * ants * problem->max_colors * problem->nof_vertices));            //[ants][max_colors][vertices]
    gpuErrchk(hipMalloc((void **) &d_probb, sizeof(double) * ants * problem->max_colors));                                    //[ants][max_colors]
    gpuErrchk(hipMalloc((void **) &d_probb_totalsum, sizeof(double) * ants));                                                 //[ants]
    gpuErrchk(hipMalloc((void **) &d_nof_confl_edges, sizeof(int) * ants));                                                   //[ants]
    gpuErrchk(hipMalloc((void **) &d_nof_confl_vertices, sizeof(int) * ants));                                                //[ants]
    gpuErrchk(hipMalloc((void **) &d_confl_vertices, sizeof(int) * ants * problem->nof_vertices));                            //[ants][nof_vertices]
    gpuErrchk(hipMalloc((void **) &d_conf, sizeof(int) * ants));                                                              //[ants]
}

/**/
__global__ void ant_fixed_k_update_1(gcp_t* d_problem, int *d_solutions_color_of, int *d_size_color, int *d_vertices, int *d_colors, int *d_nof_confl_edges, int *d_conf){
    int blockID = blockIdx.x;
    int threadID = threadIdx.x;

    d_conf[blockID + threadID] = d_nof_confl_edges[blockID + threadID];
    d_solutions_color_of[((blockID * 8 + threadID) * d_problem->nof_vertices) + d_vertices[blockID * 8 + threadID]] = d_colors[blockID * 8 + threadID];
    d_size_color[((blockID * 8) + threadID) * d_problem->max_colors + d_colors[(blockID * 8) + threadID]]++;
}

__global__ void ant_fixed_k_update_b(gcp_t* d_problem, double *d_trail, int *d_colors, int *d_vertices, int *d_neighbors_by_color,
                                     int *d_solutions_color_of, int *d_nof_confl_edges, int *d_nof_confl_vertices, int *d_confl_vertices,
                                     int *d_size_color, int *d_vertices_sat, int *d_adj_matrix, double *d_pheromone){

    int threadID = threadIdx.x;
    int blockID = blockIdx.x;

    /* trail keeps the pheromone between a vertex and all the vertex already colored with each color */
    d_trail[(blockID * d_problem->max_colors * d_problem->nof_vertices) + (d_colors[blockID] * d_problem->nof_vertices) + threadID] += d_pheromone[d_vertices[blockID] * d_problem->nof_vertices + threadID];
    if (d_adj_matrix[(d_vertices[blockID] * d_problem->nof_vertices) + threadID]){
        /* update degree of saturation: */
        if (d_neighbors_by_color[(blockID * d_problem->nof_vertices * d_problem->max_colors) + (threadID * d_problem->max_colors) + d_colors[blockID]] == 0) {
            d_vertices_sat[(blockID * d_problem->nof_vertices) + threadID]++;
        }
        /* now <i> has a neighbor colored with <color> */
        d_neighbors_by_color[(blockID * d_problem->nof_vertices * d_problem->max_colors) + (threadID * d_problem->max_colors) + d_colors[blockID]]++;

        /* if a neighbor of <v> is colored with <color>, there is a conflicting edge between them */
        if (d_solutions_color_of[(blockID * d_problem->nof_vertices)+ threadID] == d_colors[blockID]){
            d_nof_confl_edges[blockID]++;
            if (d_confl_vertices[(blockID * d_problem->nof_vertices) + threadID] == 0) {
                d_confl_vertices[(blockID * d_problem->nof_vertices) + threadID] = 1;
                d_nof_confl_vertices[blockID]++; // if(threadID == 0)
            }
        }
    }
}

__global__ void ant_fixed_k_update_2(gcp_t* d_problem, int *d_nof_confl_edges, int *d_conf, int *d_confl_vertices, int *d_nof_confl_vertices, int *d_vertices){
    int blockID = blockIdx.x;
    int threadID = threadIdx.x;

    if (d_conf[blockID * 8 + threadID] != d_nof_confl_edges[blockID + threadID]) {
        if (d_confl_vertices[((blockID * 8 + threadID) * d_problem->nof_vertices) +  d_vertices[blockID * 8 + threadID]] == 0) {
            d_confl_vertices[((blockID * 8 + threadID) * d_problem->nof_vertices) +  d_vertices[blockID * 8 + threadID]] = 1;
            d_nof_confl_vertices[blockID * 8 + threadID]++;
        }
    }
}


void print_debug(int *vprint, int tam){
  int i;
  for(i = 0; i < tam; i++){
    printf("%d ", vprint[i]);
  }
  printf("\n");
}

void print_vertices_colors(int *vertices, int *colors, int tam){
    int i;
    for (i = 0; i < tam; i++) {
        printf("vertice %d <- %d color\n", vertices[i], colors[i]);
    }
    printf("\n");
}

/* Processo ant_fixed particionado em vários kernels */
__host__ void gpu_ant_fixed_k (gcp_t* d_problem, gcp_solution_t *solutions, double* d_pheromone, hiprandState* states, int cycle, int *d_adj_matrix){
    int i;
    int colored = 0;		         /* number of colored vertex */

    /* Reseta a memória do device para começar o ciclo */
    gpu_ant_fixed_k_reset(d_problem);

    /* Times */
    double vertex_time, probbs_time, colors_time, update_time, gpu_ant_fixed_k_time;

    //printf("INFO\nceil_vertices : %d\nmax_colors : %d\n ceil_colors : %d", problem->ceil_vertices, problem->max_colors, problem->ceil_colors);

    while (colored < problem->nof_vertices){
        /* Chose the vertices to be colored */
        vertex_time = current_time_secs(TIME_INITIAL, 0);

        choose_vertex<<<ants, problem->ceil_vertices, sizeof(short) * problem->ceil_vertices * 3>>>(d_problem, d_vertices, d_vertices_sat, d_solutions_color_of);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        //debugging vertices
        //gpuErrchk(hipMemcpy(vertices, d_vertices, sizeof(int) * ants, hipMemcpyDeviceToHost));
        //printf("Vertices : \n");
        //print_debug(vertices, ants);

        vertex_time = current_time_secs(TIME_FINAL, vertex_time);

        /* Calculate colors probabilities */
        probbs_time = current_time_secs(TIME_INITIAL, 0);

        calculate_probbs<<<ants, problem->max_colors>>>(d_problem, d_probb, d_trail, d_size_color, d_neighbors_by_color, d_vertices, d_probb_totalsum);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        probbs_sum<<<ants, problem->ceil_colors, sizeof(double) * problem->ceil_colors>>>(d_problem, d_probb, d_probb_totalsum);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        probbs_time = current_time_secs(TIME_FINAL, probbs_time);

        /* Choose a color to be assigned to choosen vertices */
        colors_time = current_time_secs(TIME_INITIAL, 0);

        choose_color<<<ants, problem->max_colors, sizeof(double) * problem->max_colors>>>(d_problem, d_colors, states, d_probb_totalsum, d_probb);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        colored++;

        //debugging colors
        //gpuErrchk(hipMemcpy(colors, d_colors, sizeof(int) * ants, hipMemcpyDeviceToHost));
        //printf("Colors : \n");
        //print_debug(colors, ants);

        //debugging vertice -> color
        //print_vertices_colors(vertices, colors, ants);

        colors_time = current_time_secs(TIME_FINAL, colors_time);

        /* Update informations about conflicts and saturation degree */
        update_time = current_time_secs(TIME_INITIAL, 0);

        ant_fixed_k_update_1<<<ants/8, 8>>>(d_problem, d_solutions_color_of, d_size_color, d_vertices, d_colors, d_nof_confl_edges, d_conf);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        ant_fixed_k_update_b<<<ants, problem->nof_vertices>>>(d_problem, d_trail, d_colors, d_vertices, d_neighbors_by_color, d_solutions_color_of, d_nof_confl_edges,
                                                              d_nof_confl_vertices, d_confl_vertices, d_size_color, d_vertices_sat, d_adj_matrix, d_pheromone);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        ant_fixed_k_update_2<<<ants/8, 8>>>(d_problem, d_nof_confl_edges, d_conf, d_confl_vertices, d_nof_confl_vertices, d_vertices);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        update_time = current_time_secs(TIME_FINAL, update_time);

        /* Soma os times */
        if (problem->flags & FLAG_VERBOSE){
            total_vertex += vertex_time;
            total_probbs += probbs_time;
            total_colors += colors_time;
            total_update += update_time;
        }
    }

    /* Copia as soluções geradas */
    gpuErrchk(hipMemcpy( nof_confl_edges, d_nof_confl_edges, sizeof(int) * ants, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy( nof_confl_vertices, d_nof_confl_vertices, sizeof(int) * ants, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy( solutions_color_of, d_solutions_color_of, sizeof(int) * ants * problem->nof_vertices, hipMemcpyDeviceToHost));

    for(i = 0; i < ants; i++){
        solutions[i].nof_colors = problem->max_colors;
        solutions[i].nof_uncolored_vertices = 0;
        solutions[i].total_cycles = cycle;
        solutions[i].nof_confl_edges = nof_confl_edges[i];
        solutions[i].nof_confl_vertices = nof_confl_vertices[i];
        solutions[i].color_of = (int*) malloc(sizeof(int) * problem->nof_vertices);
        memcpy(solutions[i].color_of, &solutions_color_of[i * problem->nof_vertices], sizeof(int) * problem->nof_vertices);
        solutions[i].spent_time = current_time_secs(TIME_FINAL, time_initial);
    }

    /* Reseta a memória do device para o próximo ciclo */
    //gpu_ant_fixed_k_reset();


    /* Implementar SUPER_VERBOSE_FLAG para printar a cada ciclo os tempos
    if (problem->flags & FLAG_S_VERBOSE){
        total_time += total_vertex + total_probbs + total_colors + total_update;
        printf("\n-----------------------------------------------\ngpu_ant_fixed_k total time = %lf\n", total_time);
        printf("     choose_vertex time = %lf\n     calculate_probbs time = %lf\n     choose_color time = %lf\n     update time = %lf\n     reset time  = %lf\n\n", total_vertex, total_probbs, total_colors, total_update, reset_time);
    }*/

    /*FIM*/
}

__host__ void ant_fixed_free(){
    /* Free */
    free(probb);
    free(probb_totalsum);
    free(vertices);
    free(colors);
    free(solutions_color_of);

    /* Cuda Free */
    gpuErrchk(hipFree(d_vertices));
    gpuErrchk(hipFree(d_colors));
    gpuErrchk(hipFree(d_vertices_sat));
    gpuErrchk(hipFree(d_solutions_color_of));
    gpuErrchk(hipFree(d_neighbors_by_color));
    gpuErrchk(hipFree(d_size_color));
    gpuErrchk(hipFree(d_trail));
    gpuErrchk(hipFree(d_probb));
    gpuErrchk(hipFree(d_probb_totalsum));
}

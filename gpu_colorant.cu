#include "hip/hip_runtime.h"

/***********************************************************
* Created: Sex 17 Dez 2015
*
* Author: Bruno C.P. Dala Rosa, bcesar.g6@gmail.com
* Original version: Carla N. Lintzmayer, carla0negri@gmail.com
*
*************************************************************************
* Versão GPU
* - Esta versão foi implementada a partir da versão base e tem como intuito retirar as alocações dinâmicas dentro do kernel,
*   nela as variáveis locais de cada thread se tornam variáveis globais que englobam o escopo de todas as threads.
*   O acesso a essas variáveis é controlado por uma aritmética de ponteiro que define um intervalo de memória separado em que cada thread irá trabalhar.
*   A alocação de memória foi adaptada.

*************************************************************************/

#include <stdio.h>
#include <string.h>
#include <math.h>
#include <limits.h>

#include "aco.h"
#include "tabucol.h"
#include "color.h"
#include "gpu_ant_fixed_k.cuh"

#include "util.h"
#include "merge_sort_struct.h"

/* CUDA runtime */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

/* Time vars */
double cycle_time;
double gpu_phero_var_time;

/* Sistema de métrica de tempo de cada função */
double gpu_ant_time, gpu_tabu_time, gpu_pheromone_time;
double media_choose_vertex, media_calculate_probbs, media_choose_colors, media_ant_update;

/* extern definitions <-> aco.h*/
int* d_solutions_color_of;
int* d_nof_confl_vertices;
double total_vertex, total_probbs, total_colors, total_update;
int *d_adj_matrix;
gcp_t* d_problem;

/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/* Kernel launch configuration */
int n_threads;
int coeficiente_aproveitamento;
dim3 block,grid;

/* Global data */
static double *pheromone;
static double *phero_var;
static int gap;
double device_mem_size = 0;

static aco_memory_t *memory = NULL;
static gcp_solution_t *ant_memory_remove;
static gcp_solution_t *ant_memory_insert;

//static gcp_solution_t *ant_k;
static gcp_solution_t *best_colony;
static gcp_solution_t *best_ant;

gcp_solution_t *solutions; //Array de formigas (Soluções)
gcp_solution_t *useful_ants; //Array trimado das melhores soluções. total de formigas / coeficiente_aproveitamento

/* Device pointers */
hiprandState_t *states;
double *d_pheromone;
int *d_cycle;
double *d_phero_var;
int *d_best_ant_colors;
int *d_best_colony_colors;
int d_best_colony_nof_confl;
int d_best_ant_nof_confl;

/* Device return pointers */
int *solution_color_of;

void gpu_colorant_printbanner() {
    char *schemes[] = {"All ants + Best ant + Best colony", "Best ant + Best colony", "Best ant + Best colony (gap)"};

    fprintf(problem->fileout, "GPU-COLORANT  \n");

    fprintf(problem->fileout, "-------------------------------------------------\n");
    fprintf(problem->fileout, "Graph info:\n");
    fprintf(problem->fileout, "  Number of vertices.....................: %d\n", problem->nof_vertices);
    fprintf(problem->fileout, "  Ceil number of vertices (exp 2)........: %d\n", problem->ceil_vertices);


    fprintf(problem->fileout, "-------------------------------------------------\n");
    fprintf(problem->fileout, "Parameters:\n");

    if(!(aco_info->gpuid)){
        if (!(get_flag(problem->flags, FLAG_ANTS_RATIO))) {
            fprintf(problem->fileout, "  Ants.............................: %i\n", aco_info->nants);
        }
        else {
            fprintf(problem->fileout, "  Ants.............................: %i (%i of %i - vertices)\n", aco_info->nants, aco_info->ratio, problem->nof_vertices);
        }
    } else{
        fprintf(problem->fileout, "  Threads (Ants)...................: %d\n", aco_info->nants);

        if (coeficiente_aproveitamento > 1){
            fprintf(problem->fileout,"  Using only the best %d solutions in the process.\n", n_threads/coeficiente_aproveitamento);
        } else{
            fprintf(problem->fileout,"  Using all the solutions in the process.\n");
        }
    }

    fprintf(problem->fileout, "  Alpha............................: %.2f\n", aco_info->alpha);
    fprintf(problem->fileout, "  Beta.............................: %.2f\n", aco_info->beta);
    fprintf(problem->fileout, "  Rho..............................: %.2f\n", aco_info->rho);

    fprintf(problem->fileout, "  Pheromone scheme.................: %s\n", schemes[aco_info->pheromone_scheme-1]);

    if (get_flag(problem->flags, FLAG_CHANGE_PHEROMONE_SCHEME))
      fprintf(problem->fileout, "  Change pheromone scheme after %i iterations.\n", aco_info->change_phero_scheme_iterations);

      if (get_flag(problem->flags, FLAG_MEMORY)) {
          if (!(get_flag(problem->flags, FLAG_MEMORY_RATIO))) {
             fprintf(problem->fileout, "  Memory Usage:\n\tMemory size......................: %i\n", aco_info->memory_size);
          }
          else {
             fprintf(problem->fileout, "  Memory Usage:\n\tMemory size......................: %i (%i of %i - ants)\n", aco_info->memory_size, aco_info->memory_ratio, aco_info->nants);
          }
        fprintf(problem->fileout, "\tDelta............................: %.2f\n", aco_info->delta);
      }

      if (aco_info->pheromone_scheme == PHEROMONE_SCHEME_3)
      fprintf(problem->fileout, "  Pheromone Scheme 3:\n\tGap..............................: %i\n", aco_info->gap);


      if (get_flag(problem->flags, FLAG_CHANGE_ALPHA_BETA)) {
        fprintf(problem->fileout, "  Change alpha and beta:\n\tGamma............................: %.2f\n", aco_info->gamma);
        fprintf(problem->fileout, "\tOmega............................: %.2f\n", aco_info->omega);
        fprintf(problem->fileout, "\tChange alpha and beta after %i iterations.\n", aco_info->iterations_alpha_beta);
      }
}

void colorant_malloc() {
    aco_info = (aco_t*) malloc(sizeof(aco_t));
    aco_info->pheromone_scheme = PHEROMONE_SCHEME_1;
    aco_info->change_phero_scheme_iterations = COLORANT_CHANGE_PHERO_SCHEME_ITERATIONS;
    aco_info->iterations_alpha_beta = COLORANT_ITERATIONS_ALPHA_BETA;
    aco_info->ratio           = COLORANT_ANTS;
    aco_info->alpha           = COLORANT_ALPHA;
    aco_info->beta            = COLORANT_BETA;
    aco_info->rho             = COLORANT_RHO;
    aco_info->gap             = COLORANT_GAP;
    aco_info->gamma           = COLORANT_GAMMA;
    aco_info->omega           = COLORANT_OMEGA;
    aco_info->x               = COLORANT_X;
    aco_info->y               = COLORANT_Y;
    aco_info->memory_size     = COLORANT_MEMORY_SIZE;
    aco_info->delta           = COLORANT_DELTA;

    aco_info->gpuid           = NOGPU;
    aco_info->nants           = GPU_N_THREADS;
    aco_info->n_threads       = GPU_N_THREADS;
    aco_info->aproveitamento  = GPU_APROVEITAMENTO;

}

void colorant_initialization() {
    aco_info->alpha_base = aco_info->alpha;
    aco_info->beta_base  = aco_info->beta;


    if (get_flag(problem->flags, FLAG_ANTS_RATIO)) {
        aco_info->ratio = aco_info->nants;
        aco_info->nants = (problem->nof_vertices * aco_info->nants) / 100;
    }

    if (get_flag(problem->flags, FLAG_MEMORY_RATIO)) {
          aco_info->memory_ratio = aco_info->memory_size;
          aco_info->memory_size = (aco_info->memory_size * aco_info->nants) / 100;
      aco_info->memory_size = aco_info->memory_size < 1 ? 1 : aco_info->memory_size;
    }
}

void colorant_show_solution() {
     if (get_flag(problem->flags, FLAG_CHANGE_ALPHA_BETA)) {
        fprintf(problem->fileout, "-------------------------------------------------\n");
        fprintf(problem->fileout, "Alpha.: %.2f\n", aco_info->alpha);
        fprintf(problem->fileout, "Beta..: %.2f\n", aco_info->beta);
        fprintf(problem->fileout, "Rho...: %.2f\n", aco_info->rho);
    }
}

static int memory_length() {
  aco_memory_t *item = memory;
  int length = 0;
  for ( ; item != NULL; item = item->tail, length++);
  return length;
}

static void print_memory() {
  aco_memory_t *lmemory = memory;
  int v, count = 1;
  for (; lmemory; lmemory = lmemory->tail) {
    fprintf(problem->fileout, "Item: %i\n", count++);
    fprintf(problem->fileout, "No. of conflicting edges: %d\n", lmemory->head->nof_confl_edges);
    fprintf(problem->fileout, "No. of conflicting vertices: %d\n", lmemory->head->nof_confl_vertices);
    fprintf(problem->fileout, "Color:\n");
    for (v = 0; v < problem->nof_vertices; v++)
      fprintf(problem->fileout, "%i, ", lmemory->head->color_of[v]);
    fprintf(problem->fileout, "\n\n");
  }

  fprintf(problem->fileout, "Removed:\n");
  fprintf(problem->fileout, "No. of conflicting edges: %d\n", ant_memory_remove->nof_confl_edges);
  fprintf(problem->fileout, "No. of conflicting vertices: %d\n", ant_memory_remove->nof_confl_vertices);
  fprintf(problem->fileout, "Color:\n");
  for (v = 0; v < problem->nof_vertices; v++)
    fprintf(problem->fileout, "%i, ", ant_memory_remove->color_of[v]);
  fprintf(problem->fileout, "\n\n");

}

static void insert_into_memory(gcp_solution_t *sol) {
  aco_memory_t *item = (aco_memory_t*) malloc_(sizeof(aco_memory_t*));
  aco_memory_t *last, *previous;

  item->head = sol;
  item->tail = memory;
  memory = item;

  cpy_solution(sol, ant_memory_insert);

  if (aco_info->memory_size < memory_length()) {
    last = memory->tail;
    previous = memory;
    for ( ; last && last->tail != NULL; last = last->tail, previous = previous->tail);
    previous->tail = NULL;
    cpy_solution(last->head, ant_memory_remove);
    set_flag(problem->flags, FLAG_MEMORY_REMOVE);
    free(last);
  }

  //print_memory();

}

/* Inicializa dados locais e do device e copia dados para o device */
__host__ void initialize_data() {
    int i, j;
    n_threads = aco_info->n_threads;
    coeficiente_aproveitamento = aco_info->aproveitamento;

    phero_var = (double*) malloc(problem->nof_vertices * problem->nof_vertices * sizeof(double));

    /* Alocamento de pinned memory para dados que serão transferidos para o device */
    hipError_t status = hipHostMalloc((void**)&pheromone, problem->nof_vertices * problem->nof_vertices * sizeof(double));
    if (status != hipSuccess){
        printf("Error allocating pheromone pinned host memory");
        pheromone = (double*) malloc (problem->nof_vertices *problem->nof_vertices* sizeof(double));
    }

    gpuErrchk(hipMalloc((void **) &d_pheromone, problem->nof_vertices * problem->nof_vertices * sizeof(double)));
    device_mem_size += problem->nof_vertices * problem->nof_vertices * sizeof(double);

    gpuErrchk(hipMalloc((void **) &d_problem, sizeof(gcp_t)));

    gpuErrchk(hipMalloc((void **) &d_best_colony_colors, problem->nof_vertices * sizeof(int)));

    gpuErrchk(hipMalloc((void **) &d_best_ant_colors, problem->nof_vertices * sizeof(int)));

    gpuErrchk(hipMalloc((void**) &d_adj_matrix, problem->nof_vertices * problem->nof_vertices * sizeof(int)));
    device_mem_size += problem->nof_vertices * problem->nof_vertices * sizeof(int);

    gpuErrchk(hipMemcpy(d_adj_matrix, problem->adj_matrix, problem->nof_vertices * problem->nof_vertices * sizeof(int), hipMemcpyHostToDevice));
    /*----------------------------------------------------------------------------*/

    for (i = 0; i < problem->nof_vertices; i++) {
        for (j = 0; j < problem->nof_vertices; j++) {
            pheromone[i * problem->nof_vertices + j] = 0;
            phero_var[i * problem->nof_vertices + j] = 0;
            if (!problem->adj_matrix[i * problem->nof_vertices + j]) {
                pheromone[i * problem->nof_vertices + j] = 1;
            }
        }
    }

    best_ant = (gcp_solution_t*) malloc (sizeof(gcp_solution_t));
    best_ant->color_of = (int*) malloc (sizeof(int) * problem->nof_vertices);
    best_ant->nof_confl_vertices = INT_MAX;
    best_ant->nof_colors = problem->max_colors;

    best_colony =(gcp_solution_t*) malloc(sizeof(gcp_solution_t));
    best_colony->color_of =(int*) malloc (sizeof(int) * problem->nof_vertices);
    best_colony->nof_confl_vertices = INT_MAX;
    best_colony->nof_colors = problem->max_colors;

    if (get_flag(problem->flags, FLAG_MEMORY)) {
      ant_memory_remove = (gcp_solution_t*)  malloc_(sizeof(gcp_solution_t));
      ant_memory_remove->color_of = (int*) malloc_(sizeof(int) * problem->nof_vertices);
      ant_memory_remove->nof_colors = problem->max_colors;
      ant_memory_remove->spent_time = 0;
      ant_memory_insert = (gcp_solution_t*) malloc_(sizeof(gcp_solution_t));
      ant_memory_insert->color_of = (int*) malloc_(sizeof(int) * problem->nof_vertices);
      ant_memory_insert->nof_colors = problem->max_colors;
      ant_memory_insert->spent_time = 0;

    }

    gpuErrchk(hipMalloc((void **)&states, aco_info->n_threads * sizeof(hiprandState_t)));

    gpuErrchk(hipMalloc((void **)&d_phero_var, problem->nof_vertices * problem->nof_vertices * sizeof(double)));
    gpuErrchk(hipMemset( d_phero_var, 0, problem->nof_vertices * problem->nof_vertices * sizeof(double)));

    ant_fixed_initilization(d_problem);
    copiaConstant(problem, aco_info);

    setup_cuRand<<<aco_info->n_threads,1>>>(states, problem->seed);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());

    device_mem_size += n_threads * sizeof(hiprandState_t);
    device_mem_size += sizeof(gcp_t) + sizeof(aco_t);
}


static void update_pheromone_trails_memory(void) {
 int i, j;

 for (i = 0; i < problem->nof_vertices; i++) {
   for (j = 0; j < problem->nof_vertices; j++) {

     if (!problem->adj_matrix[i * problem->nof_vertices + j]) {
       if (ant_memory_insert->color_of[i] == ant_memory_insert->color_of[j])
         pheromone[i * problem->nof_vertices + j] *=  1 + aco_info->delta;

       if ((get_flag(problem->flags, FLAG_MEMORY_REMOVE)) &&
           (ant_memory_remove->color_of[i] == ant_memory_remove->color_of[j]))
         pheromone[i * problem->nof_vertices + j] *=  1 - aco_info->delta;
     }
   }
 }
}

/* Parallel Functions to help updating pheromone */
__global__ void gpu_update_var_phero(gcp_t* d_problem, double* d_phero_var,
    int* d_solutions_color_of, int *d_nof_confl_vertices, int *d_adj_matrix){

    int i;
    int tid = threadIdx.x;
    int nof_confl_v = d_nof_confl_vertices[0];
    int padding = tid * d_problem->nof_vertices;

    //Use shared memory
    extern __shared__ int s[];
	int* s_solution = s; //nof_vertices

    s_solution[tid] = *(d_solutions_color_of + tid);
    __syncthreads();

    for (i = 0 ; i < d_problem->nof_vertices; i++) {
        if (!d_adj_matrix[padding + i] && (s_solution[i] == s_solution[tid])){
            d_phero_var[padding + i] += nof_confl_v == 0 ? 1 : 1.0/nof_confl_v;
        }
     }
}

/* Scheme 1 */
__global__ void gpu_update_pheromone_trails_colorant1(gcp_t* d_problem,
    double* d_pheromone, double* d_phero_var, int* d_ba_solutions_color_of,
    int* d_bc_solutions_color_of, int ba_nof_confl, int bc_nof_confl,
    int *d_adj_matrix, float rho){

    int i;
    int tid = threadIdx.x;
    int padding = tid * d_problem->nof_vertices;

    //Use shared memory
    extern __shared__ int s[];
	int* ba_solution = s; //nof_vertices
    int* bc_solution = &ba_solution[d_problem->nof_vertices];

    ba_solution[tid] = *(d_ba_solutions_color_of + tid);
    bc_solution[tid] = *(d_bc_solutions_color_of + tid);
    __syncthreads();

    for (i = 0 ; i < d_problem->nof_vertices; i++) {
        d_pheromone[padding + i] += d_phero_var[padding + i];
        d_pheromone[padding + i] *= rho;


        if (!d_adj_matrix[padding + i]){

            if ((ba_solution[i] == ba_solution[tid])){
                d_pheromone[padding + i] += (ba_nof_confl == 0) ? 1 : 1.0/ba_nof_confl;
            }

            if ((bc_solution[i] == bc_solution[tid])){
                d_pheromone[padding + i] += (bc_nof_confl == 0) ? 1 : 1.0/bc_nof_confl;
            }
        }
     }
}

/* Scheme 2 */
__global__ void gpu_update_pheromone_trails_colorant2(gcp_t* d_problem,
    double* d_pheromone, int* d_ba_solutions_color_of,
    int* d_bc_solutions_color_of, int ba_nof_confl, int bc_nof_confl,
    int *d_adj_matrix, float rho){

    int i;
    int tid = threadIdx.x;
    int padding = tid * d_problem->nof_vertices;

    //Use shared memory
    extern __shared__ int s[];
	int* ba_solution = s; //nof_vertices
    int* bc_solution = &ba_solution[d_problem->nof_vertices];

    ba_solution[tid] = *(d_ba_solutions_color_of + tid);
    bc_solution[tid] = *(d_bc_solutions_color_of + tid);
    __syncthreads();

    for (i = 0 ; i < d_problem->nof_vertices; i++) {
        d_pheromone[padding + i] *= rho;

        if (!d_adj_matrix[padding + i]){

            if ((ba_solution[i] == ba_solution[tid])){
                d_pheromone[padding + i] += (ba_nof_confl == 0) ? 1 : 1.0/ba_nof_confl;
            }

            if ((bc_solution[i] == bc_solution[tid])){
                d_pheromone[padding + i] += (bc_nof_confl == 0) ? 1 : 1.0/bc_nof_confl;
            }
        }
     }
}

/* Scheme 3 */
__global__ void gpu_update_pheromone_trails_colorant345(gcp_t* d_problem,
    double* d_pheromone, int* d_ba_solutions_color_of,
    int* d_bc_solutions_color_of, int ba_nof_confl, int bc_nof_confl,
    int *d_adj_matrix, int gap, float rho){

    int i;
    int tid = threadIdx.x;
    int padding = tid * d_problem->nof_vertices;

    //Use shared memory
    extern __shared__ int s[];
	int* ba_solution = s; //nof_vertices
    int* bc_solution = &ba_solution[d_problem->nof_vertices];

    ba_solution[tid] = *(d_ba_solutions_color_of + tid);
    bc_solution[tid] = *(d_bc_solutions_color_of + tid);
    __syncthreads();

    for (i = 0 ; i < d_problem->nof_vertices; i++) {
        d_pheromone[padding + i] *= rho;

        if (!d_adj_matrix[padding + i]){
            if(gap){
                if ((ba_solution[i] == ba_solution[tid])){
                    d_pheromone[padding + i] += (ba_nof_confl == 0) ? 1 : 1.0/ba_nof_confl;
                }

            } else{
                if ((bc_solution[i] == bc_solution[tid])){
                    d_pheromone[padding + i] += (bc_nof_confl == 0) ? 1 : 1.0/bc_nof_confl;
                }
            }
        }
     }
}

/* Functions to help updating pheromone */
__host__ void update_var_phero(gcp_solution_t *solution) {
    int i, j;

    for (i = 0; i < problem->nof_vertices; i++) {
        for (j = 0; j < problem->nof_vertices; j++) {
            if (!problem->adj_matrix[i * problem->nof_vertices + j] &&
               (solution->color_of[i] == solution->color_of[j])) {

                phero_var[i * problem->nof_vertices + j] += (solution->nof_confl_vertices == 0) ? 1 : 1.0/solution->nof_confl_vertices;
            }
        }
    }
}

__host__ void update_pheromone_trails_colorant1() {
    int i, j;

    for (i = 0; i < problem->nof_vertices; i++) {
        for (j = 0; j < problem->nof_vertices; j++) {
            pheromone[i * problem->nof_vertices + j] += phero_var[i * problem->nof_vertices + j];
            pheromone[i * problem->nof_vertices + j] *= aco_info->rho;

            if (!problem->adj_matrix[i * problem->nof_vertices + j]) {
                if (best_ant->color_of[i] == best_ant->color_of[j]){
                    pheromone[i * problem->nof_vertices + j] += (best_ant->nof_confl_vertices == 0) ? 1 : 1.0/best_ant->nof_confl_vertices;
                }

                if (best_colony->color_of[i] == best_colony->color_of[j]){
                    pheromone[i * problem->nof_vertices + j] += (best_colony->nof_confl_vertices == 0) ? 1 : 1.0/best_colony->nof_confl_vertices;
                }
            }
            phero_var[i * problem->nof_vertices + j] = 0;
        }
    }
}

__host__ void update_pheromone_trails_colorant2() {
    int i, j;

    for (i = 0; i < problem->nof_vertices; i++) {
        for (j = 0; j < problem->nof_vertices; j++) {
            pheromone[i * problem->nof_vertices + j] *= aco_info->rho;

            if (!problem->adj_matrix[i * problem->nof_vertices + j]) {
                if (best_ant->color_of[i] == best_ant->color_of[j]) {
                    pheromone[i * problem->nof_vertices + j] += (best_ant->nof_confl_vertices == 0) ? 1 : 1.0/best_ant->nof_confl_vertices;
                }

                if (best_colony->color_of[i] == best_colony->color_of[j]) {
                    pheromone[i * problem->nof_vertices + j] += (best_colony->nof_confl_vertices == 0) ? 1 : 1.0/best_colony->nof_confl_vertices;

                }
            }
        }
    }
}

__host__ void update_pheromone_trails_colorant345(int cycle) {
    int i, j;

    for (i = 0; i < problem->nof_vertices; i++) {
        for (j = i; j < problem->nof_vertices; j++) {
            pheromone[i * problem->nof_vertices + j] *= aco_info->rho;

            if (!problem->adj_matrix[i * problem->nof_vertices + j]) {
                if (gap) {
                    if (best_ant->color_of[i] == best_ant->color_of[j])
                    pheromone[i * problem->nof_vertices + j] += (best_ant->nof_confl_vertices == 0) ? 1 : 1.0/best_ant->nof_confl_vertices;
                }
                else {
                    if (best_colony->color_of[i] == best_colony->color_of[j])
                    pheromone[i * problem->nof_vertices + j] += (best_colony->nof_confl_vertices == 0) ? 1 : 1.0/best_colony->nof_confl_vertices;
                }
            }
            pheromone[j * problem->nof_vertices + i] = pheromone[i * problem->nof_vertices + j];
        }
    }
    gap--;
}
/* END Functions to help updating pheromone */

/* Debugging Functions */
void print_solutions(gcp_solution_t* solutions){
    int i;
    for(i = 0; i < n_threads; i++){
        printf("%d - %d\n",i, solutions[i].nof_confl_vertices);
    }
}

void print_phero_var(double* phero_var){
    int i,j;
    int count = 0;

    for (i = 0; i < problem->nof_vertices; i++) {
        for (j = 0; j < problem->nof_vertices; j++) {
            if (phero_var[i * problem->nof_vertices + j] != 0){
                printf("%.3lf  ",phero_var[i * problem->nof_vertices + j] );
                count++;
            }
        }
    }

    printf("PHERO_VAR TOTAL = %d\n", count);
}

/* construct_solutions now will run a kernel launch instead of a for loop! */
static void construct_solutions(int cycle, double *gpu_ant_time, double *gpu_phero_var_time, double *gpu_tabu_time){
    gcp_solution_t *ant_memory;
    int i;
    best_colony->nof_confl_vertices = INT_MAX;

    /* Aloca o useful_ants e o solutions_b */
    useful_ants = (gcp_solution_t*) malloc((n_threads / coeficiente_aproveitamento) * sizeof(gcp_solution_t));
    solutions = (gcp_solution_t*) malloc(sizeof(gcp_solution_t) * n_threads);

    /* Transfere o pheromone para a global memory */
    gpuErrchk(hipMemcpy(d_pheromone, pheromone, problem->nof_vertices * problem->nof_vertices * sizeof(double), hipMemcpyHostToDevice));

    *gpu_ant_time = current_time_secs(TIME_INITIAL, 0);

    /* gpu_ant_fixed_k launch */
    gpu_ant_fixed_k(d_problem, solutions, d_pheromone, states, cycle, d_adj_matrix);


    *gpu_ant_time = current_time_secs(TIME_FINAL, *gpu_ant_time); //tempo do ant_fixed_k
    //printf("\nCiclo: %d Pre-ordenação:\n",cycle);
    //print_solutions(solutions);

    /* TABU-SEARCH EM TODAS SOL  Antigos color(3 4 5) */
    if ((get_flag(problem->flags, FLAG_TABUCOL_ALL_ANTS) && (useful_ants[0].nof_confl_vertices != 0) && (tabucol_info->cycles > 0))) {
        *gpu_tabu_time = current_time_secs(TIME_INITIAL, 0);
        for(i = 0; i < n_threads / coeficiente_aproveitamento; i++){
            tabucol(solutions+i, tabucol_info->cycles, tabucol_info->tl_style);
        }

        *gpu_tabu_time = current_time_secs(TIME_FINAL, *gpu_tabu_time);
    }

    /* Ordena o array de soluções em ordem crescente de nof_confl_vertices*/
    merge(solutions,n_threads);

    /* Trima o array de soluções para o tamanho definido pelo coeficiente de aproveitamento das soluções */
    if(coeficiente_aproveitamento > 1){
        memcpy(useful_ants, solutions, (n_threads / coeficiente_aproveitamento) * sizeof(gcp_solution_t));
    } else {
        useful_ants = solutions;
    }

    if (aco_info->pheromone_scheme == PHEROMONE_SCHEME_1){
        *gpu_phero_var_time = current_time_secs(TIME_INITIAL, 0);
        for (i = 0; i < n_threads / coeficiente_aproveitamento; i++){
            //update_var_phero(useful_ants + i); // Método sequencial
            gpu_update_var_phero<<<1,problem->nof_vertices,problem->nof_vertices * sizeof(int)>>>(d_problem, d_phero_var, &d_solutions_color_of[i * problem->nof_vertices], &d_nof_confl_vertices[i], d_adj_matrix);
        }
        *gpu_phero_var_time = current_time_secs(TIME_FINAL, *gpu_phero_var_time);

        //debugging print
        gpuErrchk(hipMemcpy(phero_var, d_phero_var, problem->nof_vertices * problem->nof_vertices * sizeof(double), hipMemcpyDeviceToHost));
        //printf("PRINT phero_var\n");
        //print_phero_var(phero_var);
    }

    /* BUSCA-TABU APENAS NA MELHOR SOLUÇÃO Antigo color (1 2)*/
    if ((!(get_flag(problem->flags, FLAG_TABUCOL_ALL_ANTS))) && (useful_ants[0].nof_confl_vertices != 0) && (tabucol_info->cycles > 0)) {
        *gpu_tabu_time = current_time_secs(TIME_INITIAL, 0);

        tabucol(useful_ants, tabucol_info->cycles, tabucol_info->tl_style);
        *gpu_tabu_time = current_time_secs(TIME_FINAL, *gpu_tabu_time);
    }

    //printf("\nCiclo: %d Pos-Tabu:\n",cycle);
    //print_solutions(solutions);

    /* Best colony sempre será a formiga da posição 0 do vetor de soluções pós-sort */
    cpy_solution(useful_ants, best_colony);
    best_colony->cycles_to_best = cycle;
    best_colony->time_to_best = useful_ants[0].spent_time;

    gpuErrchk(hipMemcpy(d_best_colony_colors, best_colony->color_of, problem->nof_vertices * sizeof(int), hipMemcpyHostToDevice));
    d_best_colony_nof_confl = best_colony->nof_confl_vertices;

    if(coeficiente_aproveitamento > 1 ) free(solutions);
    free(useful_ants);

    if (get_flag(problem->flags, FLAG_MEMORY)) {
      ant_memory = (gcp_solution_t*) malloc_(sizeof(gcp_solution_t));
      ant_memory->color_of = (int*) malloc_(sizeof(int) * problem->nof_vertices);
      cpy_solution(best_colony, ant_memory);
      insert_into_memory(ant_memory);
    }
}


gcp_solution_t* gpu_colorant() {
    int cycle = 0;
    int converg = 0;
    int change = 0;

    double media_gpu_ant = 0;
    double media_uvar_phero = 0;
    double media_tabu = 0;
    double media_pheromone = 0;
    double media_cycle = 0;

    initialize_data();
    best_ant->stop_criterion = 0;

    //printf("%f\n", aco_info->delta);
    if (problem->flags & FLAG_VERBOSE){
        fprintf(problem->fileout,"Total memory used on Device: %.4lf KBytes\n\n", device_mem_size/1024.);
    }

    while (!terminate_conditions(best_ant, cycle, converg)) {
        cycle_time = current_time_secs(TIME_INITIAL, 0);
        cycle++;
        converg++;

        construct_solutions(cycle, &gpu_ant_time, &gpu_phero_var_time, &gpu_tabu_time);

        gpu_pheromone_time = current_time_secs(TIME_INITIAL, 0);
        if (best_colony->nof_confl_vertices < best_ant->nof_confl_vertices) {
            cpy_solution(best_colony, best_ant);
            best_ant->cycles_to_best = cycle;
            best_ant->time_to_best = best_colony->spent_time;
            converg = 0;
            change = 1;

            if(aco_info->gpuid){
                gpuErrchk(hipMemcpy(d_best_ant_colors, best_colony->color_of, problem->nof_vertices * sizeof(int), hipMemcpyHostToDevice));
                d_best_ant_nof_confl = d_best_colony_nof_confl;
            }
        }


        switch (aco_info->pheromone_scheme) {
            case PHEROMONE_SCHEME_1:
            //update_pheromone_trails_colorant1();

            gpu_update_pheromone_trails_colorant1<<<1,problem->nof_vertices,problem->nof_vertices * sizeof(int) * 2>>>(d_problem,
                d_pheromone, d_phero_var, d_best_ant_colors, d_best_colony_colors, best_ant->nof_confl_vertices,
                best_colony->nof_confl_vertices, d_adj_matrix, aco_info->rho);

            gpuErrchk(hipMemset( d_phero_var, 0, problem->nof_vertices * problem->nof_vertices * sizeof(double)));
            break;

            case PHEROMONE_SCHEME_2:
            //update_pheromone_trails_colorant2();

            gpu_update_pheromone_trails_colorant2<<<1,problem->nof_vertices,problem->nof_vertices * sizeof(int) * 2>>>(d_problem,
                d_pheromone, d_best_ant_colors, d_best_colony_colors, best_ant->nof_confl_vertices,
                best_colony->nof_confl_vertices, d_adj_matrix, aco_info->rho);
            break;

            case PHEROMONE_SCHEME_3:
            if (cycle % aco_info->gap == 0) gap = cycle / aco_info->gap;

            //update_pheromone_trails_colorant345(cycle);

            gpu_update_pheromone_trails_colorant345<<<1,problem->nof_vertices,problem->nof_vertices * sizeof(int) * 2>>>(d_problem,
                d_pheromone, d_best_ant_colors, d_best_colony_colors, best_ant->nof_confl_vertices,
                best_colony->nof_confl_vertices, d_adj_matrix, gap, aco_info->rho);

            gap--;
            break;
        }


        if (get_flag(problem->flags, FLAG_MEMORY)) {
            gpuErrchk(hipMemcpy(pheromone, d_pheromone, problem->nof_vertices * problem->nof_vertices * sizeof(double), hipMemcpyDeviceToHost));

            update_pheromone_trails_memory();

            gpuErrchk(hipMemcpy(d_pheromone, pheromone, problem->nof_vertices * problem->nof_vertices * sizeof(double), hipMemcpyHostToDevice));

        }

        //debugging print
        //printf("PRINT pheromone\n");
        //print_phero_var(pheromone);
        gpu_pheromone_time = current_time_secs(TIME_FINAL, gpu_pheromone_time); //tempo do update pheromone

        if (best_ant->nof_confl_vertices == 0) {
            best_ant->nof_uncolored_vertices = 0;
            best_ant->stop_criterion = STOP_BEST;
            break;
        }

        if ( get_flag(problem->flags, FLAG_CHANGE_ALPHA_BETA) && ((cycle % aco_info->iterations_alpha_beta)==0) ) {

            aco_info->gamma = change ? (1 - aco_info->omega) * aco_info->gamma:
            (1 + aco_info->omega) * aco_info->gamma;

            aco_info->alpha = aco_info->alpha_base * aco_info->gamma;
            aco_info->beta  = aco_info->beta_base  * (1 - aco_info->gamma);

            if ( (aco_info->alpha < 0) && ( aco_info->beta > 0) )
            aco_info->gamma = (1 + aco_info->omega) * aco_info->gamma;
            else
            if ( (aco_info->alpha > 0) && (aco_info->beta < 0) )
            aco_info->gamma = (1 - aco_info->omega) * aco_info->gamma;

            aco_info->alpha = aco_info->alpha_base * aco_info->gamma;
            aco_info->beta  = aco_info->beta_base  * (1 - aco_info->gamma);

            change = 0;

            //printf("2 alfa: %.2f beta:%.2f gama:%.2f omega:%.2f\n", aco_info->alpha, aco_info->beta, aco_info->gamma, aco_info->omega);
        }

        cycle_time = current_time_secs(TIME_FINAL, cycle_time);//tempo total do ciclo

        if (problem->flags & FLAG_VERBOSE) {
            fprintf(problem->fileout, "\nCycle %d - Conflicts found: %d (edges), %d (vertices)\n", cycle, best_ant->nof_confl_edges, best_ant->nof_confl_vertices);
            fprintf(problem->fileout, "Tempo do método ant_fixed_k: %lf\n", gpu_ant_time);
            fprintf(problem->fileout, "Tempo do ciclo: %lf\n", cycle_time);

        }

        /* Soma medias de tempo */
        media_gpu_ant += gpu_ant_time;
        media_uvar_phero += gpu_phero_var_time;
        media_tabu += gpu_tabu_time;
        media_pheromone += gpu_pheromone_time;
        media_cycle += cycle_time;
    }

    if (problem->flags & FLAG_VERBOSE){
        media_choose_vertex    = total_vertex    / cycle;
        media_calculate_probbs = total_probbs    / cycle;
        media_choose_colors    = total_colors    / cycle;
        media_ant_update       = total_update    / cycle;
        media_cycle            = media_cycle     / cycle;
        media_gpu_ant          = media_gpu_ant   / cycle;
        media_tabu             = media_tabu      / cycle;
        media_pheromone        = media_pheromone / cycle;

        fprintf(problem->fileout, "\n===\nMedias de tempo dos Kernels\n===\n");
        fprintf(problem->fileout, "Choose_vertex    : %lf\nCalculate_probbs : %lf\nChoose_colors    : %lf\nUpdate_info      : %lf\n", media_choose_vertex, media_calculate_probbs, media_choose_colors, media_ant_update);
        fprintf(problem->fileout, "\n===\nMedias de tempo dos métodos\n===\n");
        fprintf(problem->fileout, "Media ciclos     : %lf\nMedia gpu_ant_k  : %lf\nMedia busca tabu : %lf\nMedia feromonio  : %lf\n Media Phero_var  : %lf\n", media_cycle, media_gpu_ant, media_tabu, media_pheromone, media_uvar_phero);
    }

    best_ant->spent_time = current_time_secs(TIME_FINAL, time_initial);
    best_ant->total_cycles = cycle;
    return best_ant;
}

#include "hip/hip_runtime.h"
/***********************************************************
 * Created: Sex 17 Dez 2015
 *
 * Modificado em 2017
 * Autor: Bruno Cesar Puli Dala Rosa
 *
 * Original version: Carla N. Lintzmayer, carla0negri@gmail.com
 *
 *************************************************************************
 *
 *
 *************************************************************************/
#include <stdio.h>
#include <math.h>
#define _GNU_SOURCE
#include <getopt.h>
#include <stdlib.h>
#include <string.h>
#include "color.h"
#include "aco.h"
#include "tabucol.h"
#include "util.h"

/* Cuda helper */

static char *namefilein;
gcp_t *problem;
aco_t *aco_info;
tabucol_t *tabucol_info;

void show_help(char *nameprog) {
  printf("Usage: %s [options] <file>\n\n", nameprog);

  printf("ColorAnt options\n");
  printf("  [ -a, --alpha     ] <value>\tDefine alpha parameter as <value>. Default: %d.\n", COLORANT_ALPHA);
  printf("  [ -b, --beta      ] <value>\tDefine beta parameter as <value>. Default: %d.\n", COLORANT_BETA);
  printf("  [ -r, --rho       ] <value>\tDefine rho parameter as <value>. Default: %f.\n", COLORANT_RHO);
  printf("  [ -A, --ants                ] <value>\tDefine number of ants as <value>. In the GPU-VERSIONS they run in parallel. Default: %d.\n", GPU_N_THREADS);
  printf("  [ -R, --use_ants_ratio                   ] \tDefine number of ants as ratio of vertices. Default: FALSE.\n");
  printf("  [ -p, --pheromone-scheme                 ] <value>\tDefine the pheromone scheme. Default: %i.\n\t\t\t\t\t\t\t1: All ants + Best ant + Best colony.\n\t\t\t\t\t\t\t2: Best ant + Best colony\n\t\t\t\t\t\t\t3: Best ant + Best colony (gap).\n", COLORANT_PHEROMONE_SCHEME);
  printf("  [ -n, --change-phero-scheme-iterations   ] <value>\tDefine iterations as <value>. Default: %d without improvement.\n", COLORANT_CHANGE_PHERO_SCHEME_ITERATIONS);

  printf("   Memory Usage:\n\t[ -m, --memory-size                      ] <value>\tDefine memory size as <value>. Default: %d.\n", COLORANT_MEMORY_SIZE);
  printf("\t[ -M, --use-memory-ratio                 ] \tDefine memory size as ratio of ants. Default: FALSE.\n");
  printf("\t[ -d, --delta                            ] <value>\tDefine delta parameter as <value>. Default: %f.\n", COLORANT_DELTA);
  printf("   Pheromone Scheme 3:\n\t[ -g, --gap                              ] <value>\tDefine gap parameter as <value>. Default: %d.\n", COLORANT_GAP);
  printf("   Try to reuse color:\n\t[ -x, --x                                ] <value>\tDefine x parameter as <value>. Default: %f.\n", COLORANT_X);
  printf("\t[ -y, --y                                ] <value>\tDefine y parameter as <value>. Default: %f.\n", COLORANT_Y);

  printf("\nGPU-VERSION options\n");
  printf("  [-z, --gpu_version] Turn on the GPU-COLORANT algorithm. Default: no-gpu-version\n");
  printf("  [-q, --quotient] <value+>\tSet the n_threads/quotient of solutions that is used in the process. Default %d.\n", GPU_APROVEITAMENTO);
  //printf("  [-Z, --nof_threads] <value>  Define number of cuda threads (64,128,256,512,1024,2048,4096,8192). Default: %d.\n", GPU_N_THREADS);

  printf("   Change alpha and beta:\n\t[ -G, --gamma                            ] <value>\tDefine gamma parameter as <value>. Default: %f.\n", COLORANT_GAMMA);
  printf("\t[ -o, --omega                            ] <value>\tDefine omega parameter as <value>. Default: %f.\n", COLORANT_OMEGA);
  printf("\t[ -i, --iterations-alpha-beta            ] <value>\tDefine iterations parameter as <value>. Default: %i.\n", COLORANT_ITERATIONS_ALPHA_BETA);

  printf("\nTabucol options\n");
  printf("\t[ -t, --tabucol-cycles                   ] <value>\tDefine maximum number of cycles in local search as <value>. Default: %d.\n", TABUCOL_CYCLES);
  printf("\t[ -T, --tabucol-convergence-cycles       ] <value>\tDefine maximum number of local search cycles without improvement.Default: %d.\n", TABUCOL_CONVERGENCE_CYCLES);
  printf("\t[ -e, --reactive-scheme                  ] \t\tDefine a reactive scheme for tabu tenure. Default: dynamic scheme.\n");
  printf("\t[ -N, --change-tabucol-scheme-iterations ] <value>\tDefine iterations as <value>. Default: %d.\n", TABUCOL_CHANGE_SCHEME_ITERATIONS);
  printf("\t[ -F, --diff-tabucol-scheme-iterations   ] <value>\tDefine iterations as <value>. Default: %d.\n", TABUCOL_DIFF_SCHEME_ITERATIONS);

  printf("\t[ -u, --apply-tabucol-all-ants                 ] \tApply tabucol on all ants. Default: only on the best ant.\n");

  printf("\nCriterion Stopping\n");
  printf("\t[ -c, --cycles                           ] <value>\tDefine number of cycles.\n");
  printf("\t[ -E, --time                             ] <value> \tDefine time in seconds.\n");
  printf("\t[ -Y, --convergence-cycles               ] <value> \tDefine maximum number of local search cycles without improvement. \n");

  printf("\nGeneral options\n");
  printf("\t[ -k, --colors                           ] <value>\tDefine promote iterations as <value>. Default: vertices.\n");
  printf("\t[ -v, --verbose                          ] \t\tDisplay informations during execution.\n");
  printf("\t[ -V, --tabucol-verbose                  ] \t\tDisplay informations during execution about local search.\n");
  printf("\t[ -s, --seed                             ] <value>\tDefine <value> as the seed of rand function. Default: time\n");
  printf("\t[ -f, --output-filename                  ] <value>\tDefine the output filename. Default: stdout.\n");
  printf("\t[ -h, --help                             ] \t\tDisplay this information.\n\n");
}

void parseargs(int argc, char *argv[]) {

  extern char *optarg;
  char op;

#if defined NRAND
  unsigned long seed;
#endif
  /* Usando getopt para tratamento dos argumentos */
  struct option longopts[] = {
      {"alpha", 1, NULL, 'a'},
      {"beta", 1, NULL, 'b'},
      {"rho", 1, NULL, 'r'},
      {"ants", 1, NULL, 'A'},
      {"use-ants-ratio", 0, NULL, 'R'},
      {"pheromone-scheme", 1, NULL, 'p'},
      {"change-phero-scheme-iterations", 1, NULL, 'n'},
      {"memory-size", 1, NULL, 'm'},
      {"use-memory-mratio", 0, NULL, 'M'},
      {"delta", 1, NULL, 'd'},
      {"gap", 1, NULL, 'g'},
      {"x", 1, NULL, 'x'},
      {"y", 1, NULL, 'y'},
      {"gama", 1, NULL, 'G'},
      {"gpu_version", 2, NULL, 'z'},
      {"quotient_of_use", 2, NULL, 'q'},
      //{"gpu_threads", 1, NULL, 'Z'},
      {"omega", 1, NULL, 'o'},
      {"iterations-alpha-beta", 1, NULL, 'i'},
      {"tabucol-cycles", 1, NULL, 't'},
      {"tabucol-convergence-cycles", 1, NULL, 'T'},
      {"reative-scheme", 0, NULL, 'e'},
      {"change-tabucol-scheme-iterations", 1, NULL, 'N'},
      {"diff-tabucol-scheme-iterations", 1, NULL, 'F'},
      {"apply-tabucol-all-ants", 0, NULL, 'u'},
      {"cycles", 1, NULL, 'c'},
      {"time", 1, NULL, 'E'},
      {"convergence-cycles", 1, NULL, 'Y'},
      {"colors", 1, NULL, 'k'},
      {"verbose", 0, NULL, 'v'},
      {"tabucol-verbose", 0, NULL, 'V'},
      {"seed", 1, NULL, 's'},
      {"output-filename", 1, NULL, 'f'},
      {"help", 0, NULL, 'h'}

  };

  while ((op = getopt_long(argc, argv, "a:b:r:A:Rp:n:m:Md:g:x:y:G:o:i:zqZ:t:T:eN:F:uc:E:Y:k:vVs:f:h", longopts, NULL)) != -1) {

    switch (op) {
      case 'a':
        aco_info->alpha = atof(optarg);
        if (aco_info->alpha <= 0.0)
  	aco_info->alpha = COLORANT_ALPHA;
        break;
      case 'b':
        aco_info->beta = atof(optarg);
        if (aco_info->beta <= 0.0)
  	aco_info->beta = COLORANT_BETA;
        break;
      case 'r':
        aco_info->rho = atof(optarg);
        if (aco_info->rho <= 0.0)
  	aco_info->rho = COLORANT_RHO;
        break;
      case 'A':
        aco_info->nants = atoi(optarg);
        if (aco_info->nants < 1) aco_info->nants = COLORANT_ANTS;
        aco_info->n_threads = aco_info->nants;
        break;
      case 'R':
        set_flag(problem->flags, FLAG_ANTS_RATIO);
        break;
      case 'p':
        aco_info->pheromone_scheme = atoi(optarg);
        if ((aco_info->pheromone_scheme < PHEROMONE_SCHEME_1) || (aco_info->pheromone_scheme > PHEROMONE_SCHEME_3))
  	aco_info->pheromone_scheme = PHEROMONE_SCHEME_1;
        break;
      case 'n':
        aco_info->change_phero_scheme_iterations = atoi(optarg);
        if (aco_info->change_phero_scheme_iterations < 1)
  	aco_info->change_phero_scheme_iterations = COLORANT_CHANGE_PHERO_SCHEME_ITERATIONS;
        set_flag(problem->flags, FLAG_CHANGE_PHEROMONE_SCHEME);
        break;
      case 'm':
        aco_info->memory_size = atoi(optarg);
        if (aco_info->memory_size < 1)
  	aco_info->memory_size = COLORANT_MEMORY_SIZE;
        set_flag(problem->flags, FLAG_MEMORY);
        break;
      case 'M':
        set_flag(problem->flags, FLAG_MEMORY);
        set_flag(problem->flags, FLAG_MEMORY_RATIO);
        break;
      case 'd':
        aco_info->delta = atof(optarg);
        if (aco_info->delta < 0.0)
  	aco_info->delta = COLORANT_DELTA;
        set_flag(problem->flags, FLAG_MEMORY);
        break;
      case 'g':
        aco_info->gap = atoi(optarg);
        if (aco_info->gap < 1)
  	aco_info->gap = COLORANT_GAP;
        break;

      case 'x':
        aco_info->x = atof(optarg);
        if (aco_info->x < 1.0)
  	aco_info->x = COLORANT_X;
        set_flag(problem->flags, FLAG_REUSE_COLOR);
        break;
      case 'y':
        aco_info->y = atof(optarg);
        if (aco_info->y < 1.0)
  	aco_info->y = COLORANT_Y;
        set_flag(problem->flags, FLAG_REUSE_COLOR);
        break;
      case 'G':
        aco_info->gamma = atof(optarg);
        if (aco_info->gamma < 1.0)
  	aco_info->gamma = COLORANT_GAMMA;
        set_flag(problem->flags, FLAG_CHANGE_ALPHA_BETA);
        break;
      case 'o':
        aco_info->omega = atof(optarg);
        if (aco_info->omega < 1.0)
  	aco_info->omega = COLORANT_OMEGA;
        set_flag(problem->flags, FLAG_CHANGE_ALPHA_BETA);
        break;
      case 'i':
        aco_info->iterations_alpha_beta = atoi(optarg);
        if (aco_info->iterations_alpha_beta < 1)
  	aco_info->iterations_alpha_beta = COLORANT_ITERATIONS_ALPHA_BETA;
        set_flag(problem->flags, FLAG_CHANGE_ALPHA_BETA);
        break;

        case 'z':
        aco_info->gpuid = 1;
        break;

        case 'q':
        aco_info->aproveitamento = atoi(optarg);
        break;

        /*case 'Z':
        aco_info->n_threads = atoi(optarg);
        break;*/

      case 't':
        tabucol_info->cycles = atoi(optarg);
        if (tabucol_info->cycles < 1)
  	tabucol_info->cycles = TABUCOL_CYCLES;
        break;
      case 'T':
        tabucol_info->convergence_cycles = atoi(optarg);
        if (tabucol_info->convergence_cycles < 1)
  	tabucol_info->convergence_cycles = TABUCOL_CONVERGENCE_CYCLES;
        set_flag(problem->flags, FLAG_TABUCOL_CONV);
        break;
      case 'e':
        tabucol_info->tl_style = TABUCOL_REACTIVE;
        break;
      case 'N':
        tabucol_info->change_scheme_iterations = atoi(optarg);
        if (tabucol_info->change_scheme_iterations < 1)
  	tabucol_info->change_scheme_iterations = TABUCOL_CHANGE_SCHEME_ITERATIONS;
        set_flag(problem->flags, FLAG_CHANGE_TABUCOL_SCHEME);
        break;
      case 'F':
        tabucol_info->diff_scheme_iterations = atoi(optarg);
        if (tabucol_info->diff_scheme_iterations < 1)
  	tabucol_info->diff_scheme_iterations = TABUCOL_DIFF_SCHEME_ITERATIONS;
        set_flag(problem->flags, FLAG_DIFF_TABUCOL_SCHEME);
        break;
      case 'u':
        set_flag(problem->flags, FLAG_TABUCOL_ALL_ANTS);
        break;
      case 'c':
        problem->max_cycles = atoi(optarg);
        if (problem->max_cycles < 1)
  	problem->max_cycles = DEFAULT_CYCLES;
        set_flag(problem->flags, FLAG_CYCLE);
        break;
      case 'E':
        problem->max_time = atof(optarg);
        if (problem->max_time < 1.0)
  	problem->max_time = DEFAULT_TIME;
        set_flag(problem->flags, FLAG_TIME);
        break;
      case 'Y':
        problem->max_cyc_converg = atoi(optarg);
        if (problem->max_cyc_converg < 1)
  	problem->max_cyc_converg = DEFAULT_CONVERGENCE_CYCLES;
        set_flag(problem->flags, FLAG_CONV);
        break;
      case 'k':
          /*ceil_colors*/
          if (problem->max_colors <= 32){
              problem->ceil_colors = 32;
          } else if (problem->max_colors <= 64){
              problem->ceil_colors = 64;
          } else {
              problem->ceil_colors = 128;
          }

        problem->max_colors = atoi(optarg);
        set_flag(problem->flags, FLAG_COLOR);
        break;

      case 'v':
        set_flag(problem->flags, FLAG_VERBOSE);
        break;
      case 'V':
        set_flag(problem->flags, FLAG_TABUCOL_VERBOSE);
        break;
      case 's':
  #if defined LRAND
        problem->seed = atol(optarg);
  #elif defined NRAND
        seed = atol(optarg);
        memcpy(problem->seed, &seed, sizeof(unsigned short)*3);
  #endif
        set_flag(problem->flags, FLAG_SEED);
        fprintf(stdout, "Sem imprimir flags (%i), está gerando semente ao invés de pegar a passada por parâmetro!!\nSei lá o que está acontecendo!!\nCom este print funciona, então vai assim!!!!\n", problem->flags);
        break;
      case 'f':
        problem->fileout = fopen(optarg, "w");
        break;
      case 'h':
        show_help(argv[0]);
        exit(0);

      }
  }


  /* O único argumento não capturado acima é o nome do arquivo de entrada,
   * se existir */
  if (optind < argc) {
    namefilein =(char*) malloc(sizeof(char) * strlen(argv[optind])+1);
    strcpy(namefilein, argv[optind++]);

    /* verificar se foi passado algum argumento a mais */
    if (optind < argc) {
      printf("error: invalid argument. Use '-h'\n");
      exit(0);
    }
  }
  else {
    printf("error: no input files\n");
    exit(0);
  }

}

void initialization(void){
      FILE *in;

      int i, j, vi, vj;
      char f, t[50];

      in = fopen(namefilein, "r");
      if (!in) {
        printf("error: no input files\n");
        exit(0);
      }

      /* Ignoring initial informations */
      while ((j = fscanf(in, "%c", &f)) && f != 'p') {
        while (f != '\n') {
          j = fscanf(in, "%c", &f);
        }
      }

      j = fscanf(in, "%s %d %d\n", t, &problem->nof_vertices, &problem->nof_edges);
      problem->degree = (int*) malloc(sizeof(int) * problem->nof_vertices);
      if (get_flag(problem->flags, FLAG_ADJ_MATRIX)) {
        problem->adj_matrix = (int*) malloc(sizeof(int) * problem->nof_vertices * problem->nof_vertices);
      }
      if (get_flag(problem->flags, FLAG_ADJ_LIST)) {
        problem->adj_list = (int**) malloc(sizeof(int*) * problem->nof_vertices);
      }

      for (i = 0; i < problem->nof_vertices; i++) {

        /*if (get_flag(problem->flags, FLAG_ADJ_MATRIX)) {
          problem->adj_matrix[i] = (char*) malloc(sizeof(char) * problem->nof_vertices);
      }*/
        if (get_flag(problem->flags, FLAG_ADJ_LIST)) {
          problem->adj_list[i] = (int*) malloc(sizeof(int) * (problem->nof_edges+1));
        }

        for (j = 0; j < problem->nof_vertices; j++) {

          if (get_flag(problem->flags, FLAG_ADJ_MATRIX)) {
    	problem->adj_matrix[i * problem->nof_vertices + j] = 0;
          }
          if (get_flag(problem->flags, FLAG_ADJ_LIST)) {
    	problem->adj_list[i][j] = 0;
          }
        }

        if (get_flag(problem->flags, FLAG_ADJ_LIST)) {
          problem->adj_list[i][problem->nof_vertices] = 0;
        }

        problem->degree[i] = 0;
      }

      for (i = 0; i < problem->nof_edges; i++) {
        j = fscanf(in, "%c %d %d\n", &f, &vi, &vj);

        if (get_flag(problem->flags, FLAG_ADJ_MATRIX)) {
          problem->adj_matrix[((vi-1) * problem->nof_vertices) + (vj-1)] = 1;    //[vi-1][vj-1]
          problem->adj_matrix[((vj-1) * problem->nof_vertices) + (vi-1)] = 1;    //[vj-1][vi-1]
        }

        if (get_flag(problem->flags, FLAG_ADJ_LIST)) {
          problem->adj_list[vi-1][0]++;
          problem->adj_list[vi-1][problem->adj_list[vi-1][0]] = vj-1;
          problem->adj_list[vj-1][0]++;
          problem->adj_list[vj-1][problem->adj_list[vj-1][0]] = vi-1;
        }

        problem->degree[vi-1]++;
        problem->degree[vj-1]++;
      }

      fclose(in);

      /*ceil_vertices*/
      if(problem->nof_vertices > 1024){

          problem->ceil_vertices = 4096;
      } else{

          problem->ceil_vertices = problem->nof_vertices <= 512 ? 512 : 1024;
      }

}


void printbanner(void) {

#if ! defined TABUCOL
  fprintf(problem->fileout, "-------------------------------------------------\n");
#endif

  if (aco_info->gpuid) gpu_colorant_printbanner();

#if defined COLORANT || defined HCA || defined TABUCOL
  fprintf(problem->fileout, "-------------------------------------------------\n");
  tabucol_printbanner();
#endif

  fprintf(problem->fileout, "-------------------------------------------------\n");
  fprintf(problem->fileout, "GENERAL Options\n");
  fprintf(problem->fileout, "-------------------------------------------------\n");
#if ! defined ANTCOL
  fprintf(problem->fileout, "  Max colors...........: %i\n", problem->max_colors);
  fprintf(problem->fileout, "  Ceil colors (exp 2)..: %i\n", problem->ceil_colors);
#endif
#if defined LRAND
  fprintf(problem->fileout, "  Seed..........................: %lu (lrand)\n", problem->seed);
#elif defined NRAND
  fprintf(problem->fileout, "  Seed..........................: %lu (nrand)\n", print_seed(problem->seed));
#endif
  if (get_flag(problem->flags, FLAG_TIME)) fprintf(problem->fileout, "  Max time......................: %lf\n", problem->max_time);
  if (get_flag(problem->flags, FLAG_CYCLE)) fprintf(problem->fileout, "  Max cycles....................: %d\n", problem->max_cycles);
  if (get_flag(problem->flags, FLAG_CONV)) fprintf(problem->fileout, "  Max cycles without improvement: %d\n", problem->max_cyc_converg);
  if (problem->flags & FLAG_VERBOSE)
    fprintf(problem->fileout, "  Running on Verbose mode.\n");
#if ! defined TABUCOL
  if (problem->flags & FLAG_TABUCOL_VERBOSE )
    fprintf(problem->fileout, "  Running Tabu Search on Verbose mode.\n");
#endif
  fprintf(problem->fileout, "-------------------------------------------------\n");

}

void test_map(gcp_solution_t *solution) {
  int i, j, n;
  int confs = 0;
  for (i = 0; i < problem->nof_vertices; i++) {
    //printf("color of %d: %d\n", i+1, solution->color_of[i]);
    if (get_flag(problem->flags, FLAG_ADJ_MATRIX)) {
      for (j = i; j < problem->nof_vertices; j++) {
	if (problem->adj_matrix[i * problem->nof_vertices + j] &&
	    solution->color_of[i] == solution->color_of[j]) {
	  //printf("ERROR!! Conflicting edge %d--%d \n", i+1, j+1);
	  confs++;
	}
      }
    }
    else {
      for (j = 1; j <= problem->adj_list[i][0]; j++) {
	n = problem->adj_list[i][j];
	if (solution->color_of[i] == solution->color_of[n]) {
	  //printf("ERROR!! Conflicting edge %d--%d \n", i+1, n+1);
	  confs++;
	}
      }
    }
  }
  if (confs != solution->nof_confl_edges) {
    fprintf(problem->fileout, "ERROR!! Confl edges = %d; Calculated = %d\n", confs, solution->nof_confl_edges);
  }
}

void cpy_solution(gcp_solution_t *src, gcp_solution_t *dst) {

  int i, j;
  if (get_flag(problem->flags, FLAG_S_ASSIGN)) {
    for (i = 0; i < problem->nof_vertices; i++) {
      dst->color_of[i] = src->color_of[i];
    }
  }
  if (get_flag(problem->flags, FLAG_S_PARTITION)) {
    for (i = 0; i <= problem->nof_vertices; i++) {
      for (j = 0; j < problem->max_colors; j++)
	dst->class_color[j][i] = src->class_color[j][i];
    }
  }

  dst->spent_time	            = src->spent_time;
  dst->time_to_best	            = src->time_to_best;
  dst->total_cycles	            = src->total_cycles;
  dst->cycles_to_best	      	= src->cycles_to_best;
  dst->nof_colors		        = src->nof_colors;
  dst->nof_confl_edges	     	= src->nof_confl_edges;
  dst->nof_confl_vertices	    = src->nof_confl_vertices;
  dst->nof_uncolored_vertices	= src->nof_uncolored_vertices;
  dst->stop_criterion		    = src->stop_criterion;

}

void show_solution(gcp_solution_t *solution) {
  fprintf(problem->fileout, "\n-------------------------------------------------\n");
  fprintf(problem->fileout, "SOLUTION:\n");
  fprintf(problem->fileout, "-------------------------------------------------\n");
  fprintf(problem->fileout, "No. of colors utilized: %d\n", solution->nof_colors);
  fprintf(problem->fileout, "No. of conflicting edges: %d\n", solution->nof_confl_edges);
  fprintf(problem->fileout, "No. of conflicting vertices: %d\n", solution->nof_confl_vertices);
  fprintf(problem->fileout, "No. of uncolored vertices: %d\n", solution->nof_uncolored_vertices);
  fprintf(problem->fileout, "Real time: %lf\n", problem->real_time);
  fprintf(problem->fileout, "Spent time: %lf\n", solution->spent_time);
#if ! defined CONSTRKGCP
  fprintf(problem->fileout, "Time to the best: %lf\n", solution->time_to_best);
  fprintf(problem->fileout, "Total of cycles: %d\n", solution->total_cycles);
  fprintf(problem->fileout, "Cycles to the best: %d\n", solution->cycles_to_best);
  fprintf(problem->fileout, "Stop criterion: %d\n", solution->stop_criterion);
#endif

#if defined COLORANT
  colorant_show_solution();
#elif defined ANTCOL
  antcol_show_solution();
#elif defined KANTCOL
  kantcol_show_solution();
#elif defined HCA
  hca_show_solution();
#elif defined TABUCOL
  tabucol_show_solution();
#elif defined CONSTRKGCP
  constr_kgcp_show_solution();
#endif

  fprintf(problem->fileout, "-------------------------------------------------\n");
  test_map(solution);
}

/* Essa função nunca é chamada? */
gcp_solution_t* init_solution(void) {
  int i;
  gcp_solution_t *solution;

  solution =(gcp_solution_t*) malloc(sizeof(gcp_solution_t));

  if (get_flag(problem->flags, FLAG_S_ASSIGN)) {
    solution->color_of =(int*) malloc(sizeof(int) * problem->nof_vertices);
  }
  if (get_flag(problem->flags, FLAG_S_PARTITION)) {
    solution->class_color =(int**) malloc(sizeof(int*) * problem->max_colors);
    for (i = 0; i < problem->max_colors; i++) {
      solution->class_color[i] =(int*) malloc(sizeof(int) * (problem->nof_vertices+1));
    }
  }

  solution->nof_colors = 0;
  solution->total_cycles = 0;
  solution->cycles_to_best = 0;
  solution->nof_confl_edges = 0;
  solution->nof_confl_vertices = 0;
  solution->nof_uncolored_vertices = 0;
  solution->stop_criterion = -1;

  return solution;
}

gcp_solution_t* find_solution() {

  gcp_solution_t* sol = NULL;

  if(aco_info->gpuid) sol = gpu_colorant();

  return sol;
}

int terminate_conditions(gcp_solution_t *solution, int cycle, int converg) {

  if (get_flag(problem->flags, FLAG_CONV) &&
      converg >= problem->max_cyc_converg) {
    solution->stop_criterion = STOP_CONV;
    return TRUE;
  }
  else if (get_flag(problem->flags, FLAG_CYCLE) &&
	   cycle >= problem->max_cycles) {
    solution->stop_criterion = STOP_CYCLES;
    return TRUE;
  }
  else if (get_flag(problem->flags, FLAG_TIME) &&
	   current_time_secs(TIME_FINAL, time_initial) >= problem->max_time) {
    solution->stop_criterion = STOP_TIME;
    return TRUE;
  }
  return FALSE;

}

int main(int argc, char *argv[]) {
  gcp_solution_t *results;

  #if defined NRAND
    unsigned long int seed;
    int x;
  #endif

  time_initial = current_time_secs(TIME_INITIAL, 0);

  problem = (gcp_t*) malloc(sizeof(gcp_t));
  init_flag(problem->flags);
  problem->nof_vertices = 0;
  problem->nof_edges = 0;
  problem->max_cycles = 0;
  problem->max_cyc_converg = 0;
  problem->max_time = 0;
  problem->max_colors = 0;
  problem->flags = 0;
  problem->degree = 0;
  problem->adj_matrix = 0;
  problem->adj_list = 0;
  problem->fileout = stdout;
#if defined LRAND
  problem->seed = 0;
#endif
#if defined NRAND
  for (x=0; x<3; x++)
      problem->seed[x] = 0;
#endif

colorant_malloc();
tabucol_malloc();

parseargs(argc, argv);


if (!(get_flag(problem->flags, FLAG_CYCLE)) && !(get_flag(problem->flags, FLAG_TIME))) {
   printf("You need to set the stop criterion.\n");
   exit(0);
}


set_flag(problem->flags, FLAG_S_ASSIGN);
set_flag(problem->flags, FLAG_ADJ_MATRIX);


if(aco_info->gpuid){
    if ((aco_info->aproveitamento == 0 || aco_info->aproveitamento == 1)){
        aco_info->aproveitamento = 1;
    }
    if(!(aco_info->n_threads == 16 || aco_info->n_threads == 64 || aco_info->n_threads == 128 || aco_info->n_threads == 256 || aco_info->n_threads == 512 || aco_info->n_threads == 1024 || aco_info->n_threads == 2048 || aco_info->n_threads ==  4096 || aco_info->n_threads == 8192)){
        printf("Not allowed number of threads, enter 16, 64, 128, 256, 512, 1024, 2048, 4096 or 8192\nAborting.\n");
        exit(0);
    }
}

initialization();

colorant_initialization();
tabucol_initialization();


  if (!(get_flag(problem->flags, FLAG_COLOR))) {
    problem->max_colors = problem->nof_vertices;
  }

  if (!(get_flag(problem->flags, FLAG_SEED))) {
#if defined LRAND
    problem->seed = create_seed();
#endif
#if defined NRAND
    seed = create_seed();
    memcpy(problem->seed, &seed, sizeof(unsigned short)*3);
#endif
    set_flag(problem->flags, FLAG_SEED);
  }

#if defined LRAND
  srand48_r(problem->seed, &problem->buffer);
#endif
#if defined NRAND
  seed48_r(problem->seed, &problem->buffer);
#endif

  printbanner();

 /* if(aco_info->gpuid){
      int devID;
      hipDeviceProp_t props;

      // This will pick the best possible CUDA capable device
      devID = findCudaDevice(argc, (const char **)argv);

      //Get GPU information
      checkCudaErrors(hipGetDevice(&devID));
      checkCudaErrors(hipGetDeviceProperties(&props, devID));
  }*/

  results = find_solution();

  problem->real_time = current_time_secs(TIME_FINAL, time_initial);

  show_solution(results);

  fclose(problem->fileout);

  return 0;

}
